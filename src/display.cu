#include "hip/hip_runtime.h"
#include "../include/display.cuh"
#include "../include/vector3d.cuh"
#include "../include/constants.hpp"

#include <hip/hip_runtime.h>
#include <>
#include "../include/display.hpp"

namespace graphics {
__global__ void transformVerticesKernel(Vector3D* vertices, Vector2D* projectedVertices, int size, Vector3D rotation, Vector3D camera) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        Vector3D vertex = vertices[idx];

        // Rotate the vertex
        cudaRotate(&vertex, rotation.x, rotation.y, rotation.z);

        // Translate the vertex
        cudaTranslate(&vertex, camera.x, camera.y, -camera.z);

        // Project the transformed vertex
        cudaProject(&vertex, &projectedVertices[idx]);
    }
}
void Display::InitalizeCuda() {
  hipMalloc((void**)&d_vertices, NUM_VERTICES * sizeof(Vector3D));
  hipMalloc((void**)&d_projectedVertices, NUM_VERTICES * sizeof(Vector2D));
}
void Display::FreeCuda() {
    hipFree(d_vertices);
    hipFree(d_projectedVertices);
}
void Display::LaunchCuda() {
    // Copy vertices to device
    hipMemcpy(d_vertices, vertices.data(), NUM_VERTICES * sizeof(Vector3D), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (NUM_VERTICES + threadsPerBlock - 1) / threadsPerBlock;
    transformVerticesKernel<<<blocksPerGrid, threadsPerBlock>>>(d_vertices, d_projectedVertices, NUM_VERTICES, rotation, camera);

    // Copy projected vertices back to host
    hipMemcpy(projectedVertices.data(), d_projectedVertices, NUM_VERTICES * sizeof(Vector2D), hipMemcpyDeviceToHost);

    // Synchronize
    hipDeviceSynchronize();
}
} // namespace graphics
