#include "../include/display.cuh"

#include <hip/hip_runtime.h>
#include <>
#include "../include/display.hpp"

namespace graphics {
void Display::InitalizeCuda() {
  hipMalloc((void**)&d_frameBuffer, displayMode.w * displayMode.h * sizeof(uint32_t));
}
void Display::FreeCuda() {
    hipFree(d_frameBuffer);
}
} // namespace graphics
